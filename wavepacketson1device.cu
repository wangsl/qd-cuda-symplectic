
#include <iostream>
#include <hip/hip_runtime_api.h>

#include "wavepacketson1device.h"
#include "cudaUtils.h"
#include "matlabUtils.h"
#include "matlabData.h"

WavepacketsOnSingleDevice::
WavepacketsOnSingleDevice(const int device_index_,
			  const int omega_start_,
			  const int n_omegas_) :
  _device_index(device_index_),
  omega_start(omega_start_),
  n_omegas(n_omegas_),
  potential_dev(0)
{ 
  insist(_device_index >= 0);
  setup_data_on_device();
}

int WavepacketsOnSingleDevice::current_device_index() const
{
  int dev_index = -1;
  checkCudaErrors(hipGetDevice(&dev_index));
  return dev_index;
}

void WavepacketsOnSingleDevice::setup_device() const
{
  if(current_device_index() != device_index()) 
    checkCudaErrors(hipSetDevice(device_index()));
}

void WavepacketsOnSingleDevice::setup_data_on_device()
{
  setup_device();

  std::cout << " Setup data on device: " << device_index() << std::endl;

  setup_potential_on_device();
}

void WavepacketsOnSingleDevice::destroy_data_on_device()
{ 
  setup_device();

  _CUDA_FREE_(potential_dev);
}

void WavepacketsOnSingleDevice::setup_potential_on_device()
{
  if(potential_dev) return;
  
  const double *potential = MatlabData::potential();
  insist(potential);
  
  const int &n1 = MatlabData::r1()->n;
  const int &n2 = MatlabData::r2()->n;
  const int &n_theta = MatlabData::theta()->n;
  
  checkCudaErrors(hipMalloc(&potential_dev, n1*n2*n_theta*sizeof(double)));
  insist(potential_dev);
  
  checkCudaErrors(hipMemcpyAsync(potential_dev, potential, n1*n2*n_theta*sizeof(double),
				  hipMemcpyHostToDevice));
}

