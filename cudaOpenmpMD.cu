
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "cudaOpenmpMD.h"
#include "cudaUtils.h"
#include "matlabUtils.h"
#include "matlabData.h"

#include "evolutionUtils.h"

inline static void divide_into_chunks(const int n, const int m, int *chunks)
{
  for(int i = 0; i < m; i++) chunks[i] = n/m;
  for(int i = 0; i < n-n/m*m; i++) chunks[i]++;
  int s = 0; for(int i = 0; i < m; i++) s += chunks[i];
  insist(s == n);
}

CUDAOpenmpMD::CUDAOpenmpMD() :
  _n_devices(0)
{ 
  setup_n_devices();
  setup_wavepackets_on_single_device();
  enable_peer_to_peer_access();
}

CUDAOpenmpMD::~CUDAOpenmpMD() 
{ 
  devices_memory_usage();
  destroy_wavepackets_on_single_device();
  disable_peer_to_peer_access();
  reset_devices();
}

void CUDAOpenmpMD::setup_n_devices()
{
  if(_n_devices) return;

  _n_devices = cudaUtils::n_devices();
  
  if(n_devices() == 1)
    std::cout << " There is 1 GPU card" << std::endl;
  else
    std::cout << " There are " << n_devices() << " GPU cards" << std::endl;
}

void CUDAOpenmpMD::devices_synchoronize()
{
  for(int i_dev = 0; i_dev < n_devices(); i_dev++) {
    checkCudaErrors(hipSetDevice(i_dev));
    checkCudaErrors(hipDeviceSynchronize());
  }
}

void CUDAOpenmpMD::devices_memory_usage() const
{
  for(int i_dev = 0; i_dev < n_devices(); i_dev++) {
    checkCudaErrors(hipSetDevice(i_dev));
    cudaUtils::device_memory_usage();
  }
}

void CUDAOpenmpMD::reset_devices()
{
  for(int i_dev = 0; i_dev < n_devices(); i_dev++) {
    checkCudaErrors(hipSetDevice(i_dev));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipDeviceReset());
  }
}

void CUDAOpenmpMD::setup_wavepackets_on_single_device()
{ 
  insist(n_devices() > 0);
  
  insist(wavepackets_on_single_device.size() == 0);
  wavepackets_on_single_device.resize(n_devices(), 0);

  const int &n = wavepackets_on_single_device.size();

  Vec<int> omegas(n);

  const int &omega_min = MatlabData::wavepacket_parameters()->omega_min;
  const int &omega_max = MatlabData::wavepacket_parameters()->omega_max;
  const int n_omegas = omega_max - omega_min + 1;

  divide_into_chunks(n_omegas, n, omegas);

  std::cout << " Omegas on devices: "; 
  omegas.show_in_one_line();
  
  int omega_start = 0;
  for(int i_dev = 0; i_dev < n; i_dev++) {
    
    checkCudaErrors(hipSetDevice(i_dev));
    
    const int n_omegas = omegas[i_dev];
    
    wavepackets_on_single_device[i_dev] = 
      new WavepacketsOnSingleDevice(i_dev, omega_start+omega_min, n_omegas);
    
    insist(wavepackets_on_single_device[i_dev]);

    omega_start += n_omegas;
  }

  setup_devices_neighbours();

  setup_device_work_dev_on_devices();

  devices_synchoronize();
  devices_memory_usage();
}

void CUDAOpenmpMD::destroy_wavepackets_on_single_device()
{
  const int &n = wavepackets_on_single_device.size();
  for(int i = 0; i < n; i++) {
    if(wavepackets_on_single_device[i]) { 
      delete wavepackets_on_single_device[i];
      wavepackets_on_single_device[i] = 0; 
    }
  }
  wavepackets_on_single_device.resize(0);
}

void CUDAOpenmpMD::enable_peer_to_peer_access() const
{ 
  if(n_devices() == 1) return;

  std::cout << " Enable peer to peer memory access" << std::endl;
  
  for(int i_dev = 0; i_dev < n_devices(); i_dev++) {
    for(int j_dev = i_dev+1; j_dev < n_devices(); j_dev++) {
      checkCudaErrors(hipSetDevice(i_dev));
      checkCudaErrors(hipDeviceEnablePeerAccess(j_dev, 0));
      
      checkCudaErrors(hipSetDevice(j_dev));
      checkCudaErrors(hipDeviceEnablePeerAccess(i_dev, 0));
    }
  }
}

void CUDAOpenmpMD::disable_peer_to_peer_access() const
{
  if(n_devices() == 1) return;
  
  std::cout << " Disable peer to peer memory access" << std::endl;
  
  for(int i_dev = 0; i_dev < n_devices(); i_dev++) {
    for(int j_dev = i_dev+1; j_dev < n_devices(); j_dev++) {
      checkCudaErrors(hipSetDevice(i_dev));
      checkCudaErrors(hipDeviceDisablePeerAccess(j_dev));
      
      checkCudaErrors(hipSetDevice(j_dev));
      checkCudaErrors(hipDeviceDisablePeerAccess(i_dev));
    }
  }
}

void CUDAOpenmpMD::setup_devices_neighbours() const
{
  if(n_devices() == 1) return;
  
  std::cout << " Setup devices neighbours" << std::endl;

  const int &n = wavepackets_on_single_device.size();

  wavepackets_on_single_device[0]->setup_neighbours(0, wavepackets_on_single_device[1]);
  
  for(int i = 1; i < n-1; i++) {
    wavepackets_on_single_device[i]->setup_neighbours(wavepackets_on_single_device[i-1],
						      wavepackets_on_single_device[i+1]);
  }
  
  wavepackets_on_single_device[n-1]->setup_neighbours(wavepackets_on_single_device[n-2], 0);
}

void CUDAOpenmpMD::setup_device_work_dev_on_devices() const
{
  const int &n = wavepackets_on_single_device.size();
  for(int i = 0; i < n; i++)
    wavepackets_on_single_device[i]->setup_device_work_dev_and_copy_streams_events();
}

void CUDAOpenmpMD::test()
{
  
  for(int L = 0; L < MatlabData::time()->total_steps; L++) {

    std::cout << "\n Step: " << L+1 << ", " << time_now() << std::endl;

    checkCudaErrors(hipProfilerStart());
    
    omp_set_num_threads(n_devices());

    for(int l = 0; l < 6; l++) {
#pragma omp parallel for default(shared)
      for(int i_dev = 0; i_dev < n_devices(); i_dev++)
	wavepackets_on_single_device[i_dev]->test_parallel();
    }
    
    checkCudaErrors(hipProfilerStop());
    
    devices_synchoronize();

    for(int i_dev = 0; i_dev < n_devices(); i_dev++)
      wavepackets_on_single_device[i_dev]->test_serial();
  }

  std::cout << std::endl;
}
